
#include <hip/hip_runtime.h>
__global__ void ExecuteFrame(float *a, float *b, float *c)
{
	int index = threadIdx.x;
	c[index] = a[index] + b[index];
}
