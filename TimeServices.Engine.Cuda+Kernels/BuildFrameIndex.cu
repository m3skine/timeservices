
#include <hip/hip_runtime.h>
__global__ void BuildFrameIndex(float *a, float *b, float *c)
{
	int index = threadIdx.x;
	c[index] = a[index] + b[index];
}
